
#include <iostream>
#include <hip/hip_runtime.h>
#include <cstring>

#define CHECK_ERR checkErr(__FILE__,__LINE__)

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, const char* const func, const char* const file,
           const int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        // We don't exit when we encounter CUDA errors in this example.
        // std::exit(EXIT_FAILURE);
    }
}

std::string PrevFile = "";
int PrevLine = 0;

void checkErr(const std::string &File, int Line) {
#ifndef NDEBUG
      hipError_t Error = hipGetLastError();
      if (Error != hipSuccess) {
        std::cout << std::endl << File
                  << ", line " << Line
                  << ": " << hipGetErrorString(Error)
                  << " (" << Error << ")"
                  << std::endl;

        if (PrevLine > 0)
          std::cout << "Previous CUDA call:" << std::endl
                    << PrevFile << ", line " << PrevLine << std::endl;
        throw;
      }
      PrevFile = File;
      PrevLine = Line;
#endif
}

// Dense x Dense Kernel
__global__ void 
__launch_bounds__(64)
 kernel_sgemm_NT_T_m56_n9_k9_lda56_ldb9_ldc56_alpha_1_beta_1_sss_aacfffd(const float * A, int A_extraOffset, const float * B, int B_extraOffset, float * C, int C_extraOffset, unsigned numElements, unsigned* flags) {
  unsigned batchID = (threadIdx.y + blockDim.y * blockIdx.x);
  if (batchID < numElements) {
    bool isFlagsProvided = (flags != nullptr);
    bool allowed = isFlagsProvided ? static_cast<bool>(flags[batchID]) : true;
    if (allowed) {
      const float * const __restrict__ glb_A = &A[batchID * 504 + 0 + A_extraOffset];
      const float * const __restrict__ glb_B = &B[batchID * 81 + 0 + B_extraOffset];
      float * const __restrict__ glb_C = &C[batchID * 504 + 0 + C_extraOffset];
      float reg0[9] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
      __shared__  __align__(8) float totalShrMem[81];
      float * localShrMem0 = &totalShrMem[81 * threadIdx.y];

      float* shrRegion0 = &localShrMem0[0];
      // using ExtendedPatchLoader
      {
        shrRegion0[threadIdx.x + 0] = glb_B[threadIdx.x + 0];
        if (threadIdx.x < 17) {
          shrRegion0[threadIdx.x + 64] = glb_B[threadIdx.x + 64];
        }
      }
      __syncthreads();
      if (threadIdx.x < 56) {
        float value;

        for (int k = 0; k < 9; ++k) {
          value = glb_A[threadIdx.x + k * 56];

          #pragma unroll
          for (int n = 0; n < 9; ++n) {
            reg0[n] += value * shrRegion0[n + 9 * k];
          }
        }
      }
      if (threadIdx.x < 56) {
        #pragma unroll
        for (int n = 0; n < 9; ++n) {
          glb_C[threadIdx.x + 56 * n] = reg0[n] + glb_C[threadIdx.x + 56 * n];
        }
      }
    }
  }
}


// Dense x Sparse Kernel
__global__ void 
__launch_bounds__(64)
 kernel_sgemm_NT_T_m56_n9_k9_lda56_ldb9_ldc56_alpha_1_beta_1_sss_6cfb6f8(const float * A, int A_extraOffset, const float * B, int B_extraOffset, float * C, int C_extraOffset, unsigned numElements, unsigned* flags) {
  unsigned batchID = (threadIdx.y + blockDim.y * blockIdx.x);
  if (batchID < numElements) {
    bool isFlagsProvided = (flags != nullptr);
    bool allowed = isFlagsProvided ? static_cast<bool>(flags[batchID]) : true;
    if (allowed) {
      const float * const __restrict__ glb_A = &A[batchID * 504 + 0 + A_extraOffset];
      const float * const __restrict__ glb_B = &B[batchID * 9 + 0 + B_extraOffset];
      float * const __restrict__ glb_C = &C[batchID * 504 + 0 + C_extraOffset];
      float reg0[9] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
      __shared__  __align__(8) float totalShrMem[9];
      float * localShrMem0 = &totalShrMem[9 * threadIdx.y];

      float* shrRegion0 = &localShrMem0[0];
      // using ExtendedPatchLoader
      {
        if (threadIdx.x < 9) {
          shrRegion0[threadIdx.x + 0] = glb_B[threadIdx.x + 0];
        }
      }
      __syncthreads();
      if (threadIdx.x < 56) {
        float value;

        value = glb_A[threadIdx.x + 0 * 56];

        // Mul begin col 0
        reg0[1] += value * shrRegion0[0];
        // Mul end col 0

        value = glb_A[threadIdx.x + 1 * 56];

        // Mul begin col 1
        reg0[1] += value * shrRegion0[1];
        // Mul end col 1

        value = glb_A[threadIdx.x + 2 * 56];

        // Mul begin col 2
        reg0[1] += value * shrRegion0[2];
        // Mul end col 2

        value = glb_A[threadIdx.x + 3 * 56];

        // Mul begin col 3
        reg0[1] += value * shrRegion0[3];
        // Mul end col 3

        value = glb_A[threadIdx.x + 4 * 56];

        // Mul begin col 4
        reg0[1] += value * shrRegion0[4];
        // Mul end col 4

        value = glb_A[threadIdx.x + 5 * 56];

        // Mul begin col 5
        reg0[1] += value * shrRegion0[5];
        // Mul end col 5

        value = glb_A[threadIdx.x + 6 * 56];

        // Mul begin col 6
        reg0[1] += value * shrRegion0[6];
        // Mul end col 6

        value = glb_A[threadIdx.x + 7 * 56];

        // Mul begin col 7
        reg0[1] += value * shrRegion0[7];
        // Mul end col 7

        value = glb_A[threadIdx.x + 8 * 56];

        // Mul begin col 8
        reg0[1] += value * shrRegion0[8];
        // Mul end col 8

      }
      if (threadIdx.x < 56) {
        #pragma unroll
        for (int n = 0; n < 9; ++n) {
          glb_C[threadIdx.x + 56 * n] = reg0[n] + glb_C[threadIdx.x + 56 * n];
        }
      }
    }
  }
}


// Dense x Dense Kernel Launcher
void sgemm_NT_T_m56_n9_k9_lda56_ldb9_ldc56_alpha_1_beta_1_sss_aacfffd(const float * A, int A_extraOffset, const float * B, int B_extraOffset, float * C, int C_extraOffset, unsigned numElements, unsigned* flags, void* streamPtr) {
  dim3 block(64, 1, 1);
  dim3 grid((numElements + 1 - 1) / 1, 1, 1);
  hipStream_t stream = (streamPtr != nullptr) ? static_cast<hipStream_t>(streamPtr) : 0;
  kernel_sgemm_NT_T_m56_n9_k9_lda56_ldb9_ldc56_alpha_1_beta_1_sss_aacfffd<<<grid,block,0,stream>>>(A, A_extraOffset, B, B_extraOffset, C, C_extraOffset, numElements, flags);
  CHECK_ERR;
}


// Dense x Sparse Kernel Launcher
void sgemm_NT_T_m56_n9_k9_lda56_ldb9_ldc56_alpha_1_beta_1_sss_6cfb6f8(const float * A, int A_extraOffset, const float * B, int B_extraOffset, float * C, int C_extraOffset, unsigned numElements, unsigned* flags, void* streamPtr) {
  dim3 block(64, 1, 1);
  dim3 grid((numElements + 1 - 1) / 1, 1, 1);
  hipStream_t stream = (streamPtr != nullptr) ? static_cast<hipStream_t>(streamPtr) : 0;
  kernel_sgemm_NT_T_m56_n9_k9_lda56_ldb9_ldc56_alpha_1_beta_1_sss_6cfb6f8<<<grid,block,0,stream>>>(A, A_extraOffset, B, B_extraOffset, C, C_extraOffset, numElements, flags);
  CHECK_ERR;
}



int main(){
  float A[56*9] = {1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.};
  float B_sparse[9] = {4., 4., 4., 4., 4., 4., 4., 4., 4.};
  float B_dense[9 * 9] = {0., 4., 0., 0., 0., 0., 0., 0., 0., 0., 4., 0., 0., 0., 0., 0., 0., 0.,
 0., 4., 0., 0., 0., 0., 0., 0., 0., 0., 4., 0., 0., 0., 0., 0., 0., 0.,
 0., 4., 0., 0., 0., 0., 0., 0., 0., 0., 4., 0., 0., 0., 0., 0., 0., 0.,
 0., 4., 0., 0., 0., 0., 0., 0., 0., 0., 4., 0., 0., 0., 0., 0., 0., 0.,
 0., 4., 0., 0., 0., 0., 0., 0., 0.};
  float C[56*9] = {0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1};
  float R1[56*9];
  float R2[56*9];

  float *A_dev = nullptr;
  float *B_sparse_dev = nullptr;
  float *B_dense_dev = nullptr;
  float *C1_dev = nullptr;
  float *C2_dev = nullptr;

  hipMalloc((void **)&A_dev, sizeof(float) * 56 * 9); CHECK_ERR;
  hipMalloc((void **)&B_sparse_dev, sizeof(float) * 9); CHECK_ERR;
  hipMalloc((void **)&B_dense_dev, sizeof(float) * 9 * 9); CHECK_ERR;
  hipMalloc((void **)&C1_dev, sizeof(float) * 56 * 9); CHECK_ERR;
  hipMalloc((void **)&C2_dev, sizeof(float) * 56 * 9); CHECK_ERR;

  hipMemcpy((void *)A_dev, (void *)A, sizeof(float) * 56 * 9, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)B_sparse_dev, (void *)B_sparse, sizeof(float) *  9, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)B_dense_dev, (void *)B_dense, sizeof(float) *  9 * 9, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)C1_dev, (void *)C, sizeof(float) * 56 * 9, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)C2_dev, (void *)C, sizeof(float) * 56 * 9, hipMemcpyHostToDevice); CHECK_ERR;

  // Dense x Dense Matrix Mult
   sgemm_NT_T_m56_n9_k9_lda56_ldb9_ldc56_alpha_1_beta_1_sss_aacfffd(A_dev, 0, B_dense_dev, 0, C1_dev, 0, 1, nullptr, nullptr);
  hipDeviceSynchronize();
  hipMemcpy(R1, C1_dev, sizeof(float)*56*9, hipMemcpyDeviceToHost);

  // Dense x Sparse Matrix Mult
   sgemm_NT_T_m56_n9_k9_lda56_ldb9_ldc56_alpha_1_beta_1_sss_6cfb6f8(A_dev, 0, B_sparse_dev, 0, C2_dev, 0, 1, nullptr, nullptr);
  hipDeviceSynchronize();
  hipMemcpy(R2, C2_dev, sizeof(float)*56*9, hipMemcpyDeviceToHost);

  std::cout << "[";
  for (int ii = 0; ii < 56*9 -1; ii++){
    std::cout << R1[ii] << ", ";
  }
  std::cout << R1[56*9 -1] << "]" << std::endl;
  std::cout << "[";
  for (int ii = 0; ii < 56*9 - 1; ii++){
    std::cout << R2[ii] << ", ";
  }
  std::cout << R2[56*9 -1] << "]" << std::endl;
  for (int i = 0; i < 56*9; i++){
    if (R1[i] != R2[i]) {
    throw std::runtime_error(" Dense x Transposed Dense and  Dense x Transposed Sparse Matrix Mismatch in Multiplication!");
    }
  }
  std::cout << " Dense x Transposed Dense and  Dense x Transposed Sparse Matrix Multiplications Match!" << std::endl;
}
