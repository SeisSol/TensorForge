
#include <iostream>
#include <hip/hip_runtime.h>
#include <cstring>

#define CHECK_ERR checkErr(__FILE__,__LINE__)

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, const char* const func, const char* const file,
           const int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        // We don't exit when we encounter CUDA errors in this example.
        // std::exit(EXIT_FAILURE);
    }
}

std::string PrevFile = "";
int PrevLine = 0;

void checkErr(const std::string &File, int Line) {
#ifndef NDEBUG
      hipError_t Error = hipGetLastError();
      if (Error != hipSuccess) {
        std::cout << std::endl << File
                  << ", line " << Line
                  << ": " << hipGetErrorString(Error)
                  << " (" << Error << ")"
                  << std::endl;

        if (PrevLine > 0)
          std::cout << "Previous CUDA call:" << std::endl
                    << PrevFile << ", line " << PrevLine << std::endl;
        throw;
      }
      PrevFile = File;
      PrevLine = Line;
#endif
}

// Dense x Dense Kernel
__global__ void 
__launch_bounds__(64)
 kernel_sgemm_T_T_m9_n9_k56_lda9_ldb9_ldc56_alpha_1_beta_1_sss_0b51dc0(const float * A, int A_extraOffset, const float * B, int B_extraOffset, float * C, int C_extraOffset, unsigned numElements, unsigned* flags) {
  unsigned batchID = (threadIdx.y + blockDim.y * blockIdx.x);
  if (batchID < numElements) {
    bool isFlagsProvided = (flags != nullptr);
    bool allowed = isFlagsProvided ? static_cast<bool>(flags[batchID]) : true;
    if (allowed) {
      const float * const __restrict__ glb_A = &A[batchID * 504 + 0 + A_extraOffset];
      const float * const __restrict__ glb_B = &B[batchID * 81 + 0 + B_extraOffset];
      float * const __restrict__ glb_C = &C[batchID * 504 + 0 + C_extraOffset];
      float reg0[9] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
      __shared__  __align__(8) float totalShrMem[612];
      float * localShrMem0 = &totalShrMem[612 * threadIdx.y];

      float* shrRegion0 = &localShrMem0[0];
      // using ExtendedTransposePatchLoader
      {
        int index;
        #pragma unroll
        for (int i = 0; i < 8; ++i) {
          index = threadIdx.x + i * 64;
          shrRegion0[(index % 9) * 59 + index / 9] = glb_A[threadIdx.x + i * 64];
        }
        if (threadIdx.x < 19) {
          index = threadIdx.x + 512;
          shrRegion0[(index % 9) * 59 + index / 9] = glb_A[threadIdx.x + 512];
        }
      }

      float* shrRegion1 = &localShrMem0[531];
      // using ExtendedPatchLoader
      {
        shrRegion1[threadIdx.x + 0] = glb_B[threadIdx.x + 0];
        if (threadIdx.x < 17) {
          shrRegion1[threadIdx.x + 64] = glb_B[threadIdx.x + 64];
        }
      }
      __syncthreads();
      if (threadIdx.x < 56) {
        float value;

        for (int k = 0; k < 9; ++k) {
          value = shrRegion0[threadIdx.x + k * 59];

          #pragma unroll
          for (int n = 0; n < 9; ++n) {
            reg0[n] += value * shrRegion1[n + 9 * k];
          }
        }
      }
      if (threadIdx.x < 56) {
        #pragma unroll
        for (int n = 0; n < 9; ++n) {
          glb_C[threadIdx.x + 56 * n] = reg0[n] + glb_C[threadIdx.x + 56 * n];
        }
      }
    }
  }
}


// Dense x Sparse Kernel
__global__ void 
__launch_bounds__(64)
 kernel_sgemm_T_T_m9_n9_k56_lda9_ldb9_ldc56_alpha_1_beta_1_sss_6452745(const float * A, int A_extraOffset, const float * B, int B_extraOffset, float * C, int C_extraOffset, unsigned numElements, unsigned* flags) {
  unsigned batchID = (threadIdx.y + blockDim.y * blockIdx.x);
  if (batchID < numElements) {
    bool isFlagsProvided = (flags != nullptr);
    bool allowed = isFlagsProvided ? static_cast<bool>(flags[batchID]) : true;
    if (allowed) {
      const float * const __restrict__ glb_A = &A[batchID * 504 + 0 + A_extraOffset];
      const float * const __restrict__ glb_B = &B[batchID * 9 + 0 + B_extraOffset];
      float * const __restrict__ glb_C = &C[batchID * 504 + 0 + C_extraOffset];
      float reg0[9] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
      __shared__  __align__(8) float totalShrMem[540];
      float * localShrMem0 = &totalShrMem[540 * threadIdx.y];

      float* shrRegion0 = &localShrMem0[0];
      // using ExtendedTransposePatchLoader
      {
        int index;
        #pragma unroll
        for (int i = 0; i < 8; ++i) {
          index = threadIdx.x + i * 64;
          shrRegion0[(index % 9) * 59 + index / 9] = glb_A[threadIdx.x + i * 64];
        }
        if (threadIdx.x < 19) {
          index = threadIdx.x + 512;
          shrRegion0[(index % 9) * 59 + index / 9] = glb_A[threadIdx.x + 512];
        }
      }

      float* shrRegion1 = &localShrMem0[531];
      // using ExtendedPatchLoader
      {
        if (threadIdx.x < 9) {
          shrRegion1[threadIdx.x + 0] = glb_B[threadIdx.x + 0];
        }
      }
      __syncthreads();
      if (threadIdx.x < 56) {
        float value;

        value = shrRegion0[threadIdx.x + 0 * 59];

        value = shrRegion0[threadIdx.x + 1 * 59];

        // Mul begin col 1
        reg0[0] += value * shrRegion1[0];
        reg0[1] += value * shrRegion1[1];
        reg0[2] += value * shrRegion1[2];
        reg0[3] += value * shrRegion1[3];
        reg0[4] += value * shrRegion1[4];
        reg0[5] += value * shrRegion1[5];
        reg0[6] += value * shrRegion1[6];
        reg0[7] += value * shrRegion1[7];
        reg0[8] += value * shrRegion1[8];
        // Mul end col 8

        value = shrRegion0[threadIdx.x + 2 * 59];

        value = shrRegion0[threadIdx.x + 3 * 59];

        value = shrRegion0[threadIdx.x + 4 * 59];

        value = shrRegion0[threadIdx.x + 5 * 59];

        value = shrRegion0[threadIdx.x + 6 * 59];

        value = shrRegion0[threadIdx.x + 7 * 59];

        value = shrRegion0[threadIdx.x + 8 * 59];

      }
      if (threadIdx.x < 56) {
        #pragma unroll
        for (int n = 0; n < 9; ++n) {
          glb_C[threadIdx.x + 56 * n] = reg0[n] + glb_C[threadIdx.x + 56 * n];
        }
      }
    }
  }
}


// Dense x Dense Kernel Launcher
void sgemm_T_T_m9_n9_k56_lda9_ldb9_ldc56_alpha_1_beta_1_sss_0b51dc0(const float * A, int A_extraOffset, const float * B, int B_extraOffset, float * C, int C_extraOffset, unsigned numElements, unsigned* flags, void* streamPtr) {
  dim3 block(64, 1, 1);
  dim3 grid((numElements + 1 - 1) / 1, 1, 1);
  hipStream_t stream = (streamPtr != nullptr) ? static_cast<hipStream_t>(streamPtr) : 0;
  kernel_sgemm_T_T_m9_n9_k56_lda9_ldb9_ldc56_alpha_1_beta_1_sss_0b51dc0<<<grid,block,0,stream>>>(A, A_extraOffset, B, B_extraOffset, C, C_extraOffset, numElements, flags);
  CHECK_ERR;
}


// Dense x Sparse Kernel Launcher
void sgemm_T_T_m9_n9_k56_lda9_ldb9_ldc56_alpha_1_beta_1_sss_6452745(const float * A, int A_extraOffset, const float * B, int B_extraOffset, float * C, int C_extraOffset, unsigned numElements, unsigned* flags, void* streamPtr) {
  dim3 block(64, 1, 1);
  dim3 grid((numElements + 1 - 1) / 1, 1, 1);
  hipStream_t stream = (streamPtr != nullptr) ? static_cast<hipStream_t>(streamPtr) : 0;
  kernel_sgemm_T_T_m9_n9_k56_lda9_ldb9_ldc56_alpha_1_beta_1_sss_6452745<<<grid,block,0,stream>>>(A, A_extraOffset, B, B_extraOffset, C, C_extraOffset, numElements, flags);
  CHECK_ERR;
}



int main(){
  // Element Matrices
  std::cout << "Instantiating core matrices" << std::endl;
  float CoreA[9*56] = {1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.};
  float CoreB_sparse[9] = {5., 5., 5., 5., 5., 5., 5., 5., 5.};
  float CoreB_dense[9 * 9] = {0., 0., 0., 0., 0., 0., 0., 0., 0., 5., 5., 5., 5., 5., 5., 5., 5., 5.,
 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0.,
 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0.,
 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0.,
 0., 0., 0., 0., 0., 0., 0., 0., 0.};
  float CoreC[56*9] = {0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1};
  
  // Buffers 
  std::cout << "Instantiating buffer matrices" << std::endl;
  float* A = new float[9*56*479958];
  float* B_dense = new float[9*9*479958];
  float* B_sparse = new float[9*479958];
  float* C = new float[56*9*479958];
  float* R1 = new float[56*9*479958];
  float* R2 = new float[56*9*479958];

  // Copy the Element Matrices N times into Element Buffers
  std::cout << "Copying core matrices to buffers" << std::endl;
  for (int i = 0; i < 479958; i++){
    std::memcpy(A + 9 * 56 * i, CoreA, 9 * 56);
    std::memcpy(B_dense + 9 * 9 * i, CoreB_dense, 9 * 9);
    std::memcpy(B_sparse + 9 * i, CoreB_sparse, 9);
    std::memcpy(C + 56 * 9 * i, CoreC, 56 * 9);
  }

  float *A_dev = nullptr;
  float *B_sparse_dev = nullptr;
  float *B_dense_dev = nullptr;
  float *C1_dev = nullptr;
  float *C2_dev = nullptr;

  std::cout << "Allocating device memory" << std::endl;
  hipMalloc((void **)&A_dev, sizeof(float) * 9 * 56 * 479958); CHECK_ERR;
  hipMalloc((void **)&B_sparse_dev, sizeof(float) * 9 * 479958); CHECK_ERR;
  hipMalloc((void **)&B_dense_dev, sizeof(float) * 9 * 9 * 479958); CHECK_ERR;
  hipMalloc((void **)&C1_dev, sizeof(float) * 56 * 9 * 479958); CHECK_ERR;
  hipMalloc((void **)&C2_dev, sizeof(float) * 56 * 9 * 479958); CHECK_ERR;

  std::cout << "Copying buffers to device" << std::endl;
  hipMemcpy((void *)A_dev, (void *)A, sizeof(float) * 9 * 56 * 479958, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)B_sparse_dev, (void *)B_sparse, sizeof(float) *  9 * 479958, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)B_dense_dev, (void *)B_dense, sizeof(float) *  9 * 9 * 479958, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)C1_dev, (void *)C, sizeof(float) * 56 * 9 * 479958, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)C2_dev, (void *)C, sizeof(float) * 56 * 9 * 479958, hipMemcpyHostToDevice); CHECK_ERR;

  // Dense x Dense Matrix Mult
  std::cout << "Calling Dense x Dense kernel" << std::endl;
  float elapsedTime = 0.0; 
  hipEvent_t startDD, stopDD;
  hipEventCreate(&startDD);
  hipEventCreate(&stopDD);
  hipEventRecord(startDD);
   sgemm_T_T_m9_n9_k56_lda9_ldb9_ldc56_alpha_1_beta_1_sss_0b51dc0(A_dev, 0, B_dense_dev, 0, C1_dev, 0, 479958, nullptr, nullptr);
  hipEventRecord(stopDD);
  hipEventSynchronize(stopDD);
  hipEventElapsedTime(&elapsedTime, startDD, stopDD);
  std::cout << "Dense x Dense kernel took " << elapsedTime << " ms" << std::endl; 
  hipDeviceSynchronize();
  hipMemcpy(R1, C1_dev, sizeof(float)*56 * 9 * 479958, hipMemcpyDeviceToHost);

  // Dense x Sparse Matrix Mult
  std::cout << "Calling Dense x Sparse kernel" << std::endl;
  elapsedTime = 0.0;
  hipEvent_t startDS, stopDS;
  hipEventCreate(&startDS);
  hipEventCreate(&stopDS);
  hipEventRecord(startDS);
   sgemm_T_T_m9_n9_k56_lda9_ldb9_ldc56_alpha_1_beta_1_sss_6452745(A_dev, 0, B_sparse_dev, 0, C2_dev, 0, 479958, nullptr, nullptr);
  hipEventRecord(stopDS);
  hipEventSynchronize(stopDS);
  hipEventElapsedTime(&elapsedTime, startDS, stopDS);
  std::cout << "Dense x Sparse kernel took " << elapsedTime << " ms" << std::endl; 
  hipDeviceSynchronize();
  hipMemcpy(R2, C2_dev, sizeof(float)*56 * 9 * 479958, hipMemcpyDeviceToHost);

  std::cout << "Freeing device memory" << std::endl;
  hipFree(A_dev);
  hipFree(B_sparse_dev);
  hipFree(B_dense_dev);
  hipFree(C1_dev);
  hipFree(C2_dev);

  std::cout << "[";
  for (int ii = 0; ii < 56*9 -1; ii++){
    std::cout << R1[ii] << ", ";
  }
  std::cout << R1[56*9 -1] << "]" << std::endl;
  std::cout << "[";
  for (int ii = 0; ii < 56*9 - 1; ii++){
    std::cout << R2[ii] << ", ";
  }
  std::cout << R2[56*9 -1] << "]" << std::endl;
  for (int el = 0; el < 479958; el++) {
    for (int i = 0; i < 56; i++){
        for (int j = 0; j < 9; j++) {
        if (std::abs(R1[i*9 + j] - R2[i*9 + j]) > 0.001) {
            throw std::runtime_error("Transposed Dense x Transposed Dense and Transposed Dense x Transposed Sparse Matrix Mismatch in Multiplication at (" + std::to_string(i) +"," + std::to_string(j) + ")\n" + 
                std::to_string(R1[i*9 + j]) + " != " + std::to_string(R2[i*9 + j]));
        }
        }
    }
  }
  std::cout << "Results Match!" << std::endl;
}
