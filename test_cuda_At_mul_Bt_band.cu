
#include <iostream>
#include <hip/hip_runtime.h>
#include <cstring>

#define CHECK_ERR checkErr(__FILE__,__LINE__)

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, const char* const func, const char* const file,
           const int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        // We don't exit when we encounter CUDA errors in this example.
        // std::exit(EXIT_FAILURE);
    }
}

std::string PrevFile = "";
int PrevLine = 0;

void checkErr(const std::string &File, int Line) {
#ifndef NDEBUG
      hipError_t Error = hipGetLastError();
      if (Error != hipSuccess) {
        std::cout << std::endl << File
                  << ", line " << Line
                  << ": " << hipGetErrorString(Error)
                  << " (" << Error << ")"
                  << std::endl;

        if (PrevLine > 0)
          std::cout << "Previous CUDA call:" << std::endl
                    << PrevFile << ", line " << PrevLine << std::endl;
        throw;
      }
      PrevFile = File;
      PrevLine = Line;
#endif
}

// Dense x Dense Kernel
__global__ void 
__launch_bounds__(64)
 kernel_sgemm_T_T_m9_n9_k56_lda9_ldb9_ldc56_alpha_1_beta_1_sss_0b51dc0(const float * A, int A_extraOffset, const float * B, int B_extraOffset, float * C, int C_extraOffset, unsigned numElements, unsigned* flags) {
  unsigned batchID = (threadIdx.y + blockDim.y * blockIdx.x);
  if (batchID < numElements) {
    bool isFlagsProvided = (flags != nullptr);
    bool allowed = isFlagsProvided ? static_cast<bool>(flags[batchID]) : true;
    if (allowed) {
      const float * const __restrict__ glb_A = &A[batchID * 504 + 0 + A_extraOffset];
      const float * const __restrict__ glb_B = &B[batchID * 81 + 0 + B_extraOffset];
      float * const __restrict__ glb_C = &C[batchID * 504 + 0 + C_extraOffset];
      float reg0[9] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
      __shared__  __align__(8) float totalShrMem[612];
      float * localShrMem0 = &totalShrMem[612 * threadIdx.y];

      float* shrRegion0 = &localShrMem0[0];
      // using ExtendedTransposePatchLoader
      {
        int index;
        #pragma unroll
        for (int i = 0; i < 8; ++i) {
          index = threadIdx.x + i * 64;
          shrRegion0[(index % 9) * 59 + index / 9] = glb_A[threadIdx.x + i * 64];
        }
        if (threadIdx.x < 19) {
          index = threadIdx.x + 512;
          shrRegion0[(index % 9) * 59 + index / 9] = glb_A[threadIdx.x + 512];
        }
      }

      float* shrRegion1 = &localShrMem0[531];
      // using ExtendedPatchLoader
      {
        shrRegion1[threadIdx.x + 0] = glb_B[threadIdx.x + 0];
        if (threadIdx.x < 17) {
          shrRegion1[threadIdx.x + 64] = glb_B[threadIdx.x + 64];
        }
      }
      __syncthreads();
      if (threadIdx.x < 56) {
        float value;

        for (int k = 0; k < 9; ++k) {
          value = shrRegion0[threadIdx.x + k * 59];

          #pragma unroll
          for (int n = 0; n < 9; ++n) {
            reg0[n] += value * shrRegion1[n + 9 * k];
          }
        }
      }
      if (threadIdx.x < 56) {
        #pragma unroll
        for (int n = 0; n < 9; ++n) {
          glb_C[threadIdx.x + 56 * n] = reg0[n] + glb_C[threadIdx.x + 56 * n];
        }
      }
    }
  }
}


// Dense x Sparse Kernel
__global__ void 
__launch_bounds__(64)
 kernel_sgemm_T_T_m9_n9_k56_lda9_ldb9_ldc56_alpha_1_beta_1_sss_2da3af1(const float * A, int A_extraOffset, const float * B, int B_extraOffset, float * C, int C_extraOffset, unsigned numElements, unsigned* flags) {
  unsigned batchID = (threadIdx.y + blockDim.y * blockIdx.x);
  if (batchID < numElements) {
    bool isFlagsProvided = (flags != nullptr);
    bool allowed = isFlagsProvided ? static_cast<bool>(flags[batchID]) : true;
    if (allowed) {
      const float * const __restrict__ glb_A = &A[batchID * 504 + 0 + A_extraOffset];
      const float * const __restrict__ glb_B = &B[batchID * 25 + 0 + B_extraOffset];
      float * const __restrict__ glb_C = &C[batchID * 504 + 0 + C_extraOffset];
      float reg0[9] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
      __shared__  __align__(8) float totalShrMem[556];
      float * localShrMem0 = &totalShrMem[556 * threadIdx.y];

      float* shrRegion0 = &localShrMem0[0];
      // using ExtendedTransposePatchLoader
      {
        int index;
        #pragma unroll
        for (int i = 0; i < 8; ++i) {
          index = threadIdx.x + i * 64;
          shrRegion0[(index % 9) * 59 + index / 9] = glb_A[threadIdx.x + i * 64];
        }
        if (threadIdx.x < 19) {
          index = threadIdx.x + 512;
          shrRegion0[(index % 9) * 59 + index / 9] = glb_A[threadIdx.x + 512];
        }
      }

      float* shrRegion1 = &localShrMem0[531];
      // using ExtendedPatchLoader
      {
        if (threadIdx.x < 25) {
          shrRegion1[threadIdx.x + 0] = glb_B[threadIdx.x + 0];
        }
      }
      __syncthreads();
      if (threadIdx.x < 56) {
        float value;

        value = shrRegion0[threadIdx.x + 0 * 59];

        // Mul begin col 0
        reg0[0] += value * shrRegion1[0];
        reg0[1] += value * shrRegion1[1];
        // Mul end col 1

        value = shrRegion0[threadIdx.x + 1 * 59];

        // Mul begin col 1
        reg0[0] += value * shrRegion1[2];
        reg0[1] += value * shrRegion1[3];
        reg0[2] += value * shrRegion1[4];
        // Mul end col 2

        value = shrRegion0[threadIdx.x + 2 * 59];

        // Mul begin col 2
        reg0[1] += value * shrRegion1[5];
        reg0[2] += value * shrRegion1[6];
        reg0[3] += value * shrRegion1[7];
        // Mul end col 3

        value = shrRegion0[threadIdx.x + 3 * 59];

        // Mul begin col 3
        reg0[2] += value * shrRegion1[8];
        reg0[3] += value * shrRegion1[9];
        reg0[4] += value * shrRegion1[10];
        // Mul end col 4

        value = shrRegion0[threadIdx.x + 4 * 59];

        // Mul begin col 4
        reg0[3] += value * shrRegion1[11];
        reg0[4] += value * shrRegion1[12];
        reg0[5] += value * shrRegion1[13];
        // Mul end col 5

        value = shrRegion0[threadIdx.x + 5 * 59];

        // Mul begin col 5
        reg0[4] += value * shrRegion1[14];
        reg0[5] += value * shrRegion1[15];
        reg0[6] += value * shrRegion1[16];
        // Mul end col 6

        value = shrRegion0[threadIdx.x + 6 * 59];

        // Mul begin col 6
        reg0[5] += value * shrRegion1[17];
        reg0[6] += value * shrRegion1[18];
        reg0[7] += value * shrRegion1[19];
        // Mul end col 7

        value = shrRegion0[threadIdx.x + 7 * 59];

        // Mul begin col 7
        reg0[6] += value * shrRegion1[20];
        reg0[7] += value * shrRegion1[21];
        reg0[8] += value * shrRegion1[22];
        // Mul end col 8

        value = shrRegion0[threadIdx.x + 8 * 59];

        // Mul begin col 8
        reg0[7] += value * shrRegion1[23];
        reg0[8] += value * shrRegion1[24];
        // Mul end col 8

      }
      if (threadIdx.x < 56) {
        #pragma unroll
        for (int n = 0; n < 9; ++n) {
          glb_C[threadIdx.x + 56 * n] = reg0[n] + glb_C[threadIdx.x + 56 * n];
        }
      }
    }
  }
}


// Dense x Dense Kernel Launcher
void sgemm_T_T_m9_n9_k56_lda9_ldb9_ldc56_alpha_1_beta_1_sss_0b51dc0(const float * A, int A_extraOffset, const float * B, int B_extraOffset, float * C, int C_extraOffset, unsigned numElements, unsigned* flags, void* streamPtr) {
  dim3 block(64, 1, 1);
  dim3 grid((numElements + 1 - 1) / 1, 1, 1);
  hipStream_t stream = (streamPtr != nullptr) ? static_cast<hipStream_t>(streamPtr) : 0;
  kernel_sgemm_T_T_m9_n9_k56_lda9_ldb9_ldc56_alpha_1_beta_1_sss_0b51dc0<<<grid,block,0,stream>>>(A, A_extraOffset, B, B_extraOffset, C, C_extraOffset, numElements, flags);
  CHECK_ERR;
}


// Dense x Sparse Kernel Launcher
void sgemm_T_T_m9_n9_k56_lda9_ldb9_ldc56_alpha_1_beta_1_sss_2da3af1(const float * A, int A_extraOffset, const float * B, int B_extraOffset, float * C, int C_extraOffset, unsigned numElements, unsigned* flags, void* streamPtr) {
  dim3 block(64, 1, 1);
  dim3 grid((numElements + 1 - 1) / 1, 1, 1);
  hipStream_t stream = (streamPtr != nullptr) ? static_cast<hipStream_t>(streamPtr) : 0;
  kernel_sgemm_T_T_m9_n9_k56_lda9_ldb9_ldc56_alpha_1_beta_1_sss_2da3af1<<<grid,block,0,stream>>>(A, A_extraOffset, B, B_extraOffset, C, C_extraOffset, numElements, flags);
  CHECK_ERR;
}



int main(){
  float A[9*56] = {1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.};
  float B_sparse[25] = {2., 3., 1., 2., 3., 1., 2., 3., 1., 2., 3., 1., 2., 3., 1., 2., 3., 1.,
 2., 3., 1., 2., 3., 1., 2.};
  float B_dense[9 * 9] = {2., 3., 0., 0., 0., 0., 0., 0., 0., 1., 2., 3., 0., 0., 0., 0., 0., 0.,
 0., 1., 2., 3., 0., 0., 0., 0., 0., 0., 0., 1., 2., 3., 0., 0., 0., 0.,
 0., 0., 0., 1., 2., 3., 0., 0., 0., 0., 0., 0., 0., 1., 2., 3., 0., 0.,
 0., 0., 0., 0., 0., 1., 2., 3., 0., 0., 0., 0., 0., 0., 0., 1., 2., 3.,
 0., 0., 0., 0., 0., 0., 0., 1., 2.};
  float C[56*9] = {0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1};
  float R1[56*9];
  float R2[56*9];

  float *A_dev = nullptr;
  float *B_sparse_dev = nullptr;
  float *B_dense_dev = nullptr;
  float *C1_dev = nullptr;
  float *C2_dev = nullptr;

  hipMalloc((void **)&A_dev, sizeof(float) * 9 * 56); CHECK_ERR;
  hipMalloc((void **)&B_sparse_dev, sizeof(float) * 25); CHECK_ERR;
  hipMalloc((void **)&B_dense_dev, sizeof(float) * 9 * 9); CHECK_ERR;
  hipMalloc((void **)&C1_dev, sizeof(float) * 56 * 9); CHECK_ERR;
  hipMalloc((void **)&C2_dev, sizeof(float) * 56 * 9); CHECK_ERR;

  hipMemcpy((void *)A_dev, (void *)A, sizeof(float) * 9 * 56, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)B_sparse_dev, (void *)B_sparse, sizeof(float) *  25, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)B_dense_dev, (void *)B_dense, sizeof(float) *  9 * 9, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)C1_dev, (void *)C, sizeof(float) * 56 * 9, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)C2_dev, (void *)C, sizeof(float) * 56 * 9, hipMemcpyHostToDevice); CHECK_ERR;

  // Dense x Dense Matrix Mult
   sgemm_T_T_m9_n9_k56_lda9_ldb9_ldc56_alpha_1_beta_1_sss_0b51dc0(A_dev, 0, B_dense_dev, 0, C1_dev, 0, 1, nullptr, nullptr);
  hipDeviceSynchronize();
  hipMemcpy(R1, C1_dev, sizeof(float)*56*9, hipMemcpyDeviceToHost);

  // Dense x Sparse Matrix Mult
   sgemm_T_T_m9_n9_k56_lda9_ldb9_ldc56_alpha_1_beta_1_sss_2da3af1(A_dev, 0, B_sparse_dev, 0, C2_dev, 0, 1, nullptr, nullptr);
  hipDeviceSynchronize();
  hipMemcpy(R2, C2_dev, sizeof(float)*56*9, hipMemcpyDeviceToHost);

  std::cout << "[";
  for (int ii = 0; ii < 56*9 -1; ii++){
    std::cout << R1[ii] << ", ";
  }
  std::cout << R1[56*9 -1] << "]" << std::endl;
  std::cout << "[";
  for (int ii = 0; ii < 56*9 - 1; ii++){
    std::cout << R2[ii] << ", ";
  }
  std::cout << R2[56*9 -1] << "]" << std::endl;
  for (int i = 0; i < 56*9; i++){
    if (R1[i] != R2[i]) {
    throw std::runtime_error("Transposed Dense x Transposed Dense and Transposed Dense x Transposed Sparse Matrix Mismatch in Multiplication!");
    }
  }
  std::cout << "Transposed Dense x Transposed Dense and Transposed Dense x Transposed Sparse Matrix Multiplications Match!" << std::endl;
}
